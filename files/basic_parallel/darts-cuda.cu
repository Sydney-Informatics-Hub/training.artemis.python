#include "hip/hip_runtime.h"
/* Compute pi in serial */

/* https://github.com/PawseySC/Intermediate-Supercomputing/tree/master/darts/cuda/c */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
static long num_trials = 10000000;

__global__ void kernel(int* Ncirc_t_device,float *randnum)
{
  int i;
  double r = 1.0; // radius of circle
  double r2 = r*r;
  double x,y;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  x=randnum[2*i];
  y=randnum[2*i+1];
  
  if ((x*x + y*y) <= r2)
      Ncirc_t_device[i]=1;
  else 
      Ncirc_t_device[i]=0;
}

int main(int argc, char **argv) {
  int i;
  long Ncirc=0;
  int *Ncirc_t_device;
  int *Ncirc_t_host;
  float *randnum;
  int threads, blocks;
  double pi;

  // Allocate an array for the random numbers in GPU memory space
  hipMalloc((void**)&randnum,(2*num_trials)*sizeof(float));

  // Generate random numbers 
  int status;
  hiprandGenerator_t randgen;
  status = hiprandCreateGenerator(&randgen, HIPRAND_RNG_PSEUDO_MRG32K3A);
  status |= hiprandSetPseudoRandomGeneratorSeed(randgen, 4294967296ULL^time(NULL));
  status |= hiprandGenerateUniform(randgen, randnum, (2*num_trials));
  status |= hiprandDestroyGenerator(randgen);  

  threads=1000;
  blocks=num_trials/threads; 

  // Allocate hit array on host
  Ncirc_t_host=(int*)malloc(num_trials*sizeof(int));
  // Allocate hit array on device
  hipMalloc((void**)&Ncirc_t_device,num_trials*sizeof(int));

  kernel <<<blocks, threads>>> (Ncirc_t_device,randnum);

  // Synchronize host and device
  hipDeviceSynchronize();

  // Copy the hit array to host
  hipMemcpy(Ncirc_t_host,Ncirc_t_device,num_trials*sizeof(int),hipMemcpyDeviceToHost);

  // Count hits 
  for(i=0; i<num_trials; i++)
    Ncirc+=Ncirc_t_host[i];

  pi = 4.0 * ((double)Ncirc)/((double)num_trials);
  
  printf("\n \t Computing pi in serial: \n");
  printf("\t For %ld trials, pi = %f\n", num_trials, pi);
  printf("\n");

  hipFree(randnum);
  hipFree(Ncirc_t_device);
  free(Ncirc_t_host);

  return 0;
}

